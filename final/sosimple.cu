#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

__global__ void kernel(void) {
}

int kernel_test()
{
  size_t available, total;
  hipMemGetInfo(&available, &total);
  printf("Memory available: %ld, Total: %ld\n", available, total);

  float *dev_values;
  size_t size = num_values * sizeof(float);

  hipMalloc((void**) &dev_values, size);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);
  kernel<<<1,1>>>();
  printf("finished \n");
  return 0;
}

